#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "argparse.h"
#include "kalman.h"

//#define DEBUG

using namespace std;

#define SCRATCH_SIZE (2 * N * N + 3 * N)

int N, days, grid_dim, block_dim;

__device__ KalmanResult kalman_cuda_update(int N, int obs,
                                           double *x, double *P,
                                           const double *prices,
                                           double *scratch) {
	auto tid = threadIdx.x;
	auto block_dim = blockDim.x;

	auto N2 = N * N;
	auto Q = DELTA / (1 - DELTA);
	auto z = prices[obs];
	auto H = scratch + 0;
	int i_H = 0;
	for(int i = 0; i < N; i++) {
		if(i == obs)
			continue;
		H[i_H] = prices[i];
		i_H++;
	}
	H[i_H] = 1;

#ifdef DEBUG
	if(obs == 1 && tid == 0) {
		printf("H\n");
		for(int i = 0; i < N; i++) {
			printf("%.2lf\t", H[i]);
		}
		printf("\n");
	}
#endif

	/////////////
	// predict //
	/////////////

	// state prediction

	auto x_apriori = x;

#ifdef DEBUG
	if(obs == 1 && tid == 0) {
		printf("x_apriori\n");
		for(int i = 0; i < N; i++) {
			printf("%.2lf\t", x_apriori[i]);
		}
		printf("\n");
	}
#endif

	// state covariance prediction

	auto P_apriori = H + N;
	for(int j = tid; j < N; j += block_dim) {
		for(int i = 0; i < N; i++) {
			auto idx = i + j * N;
			P_apriori[idx] = P[idx];
			if(i == j)
				P_apriori[idx] += Q;
		}
	}

#ifdef DEBUG
    if(obs == 1 && tid == 0) {
        printf("P_apriori\n");
        for(int i = 0; i < N; i++) {
            for(int j = 0; j < N; j++) {
                printf("%.2lf\t", P_apriori[i + j * N]);
            }
            printf("\n");
        }
    }
#endif

	///////////////
	// calculate //
	///////////////

	// innovation residual

	double y = z;
	for(int i = 0; i < N; i++) {
		y -= H[i] * x_apriori[i];
	}

#ifdef DEBUG
	if(obs == 1 && tid == 0) {
		printf("y %.2lf\n", y);
	}
#endif

	// innovation covariance

	double s = R;
	auto P_H = P_apriori + N2;
	for(int i = tid; i < N; i += block_dim) {
		double dot = 0;
		for(int j = 0; j < N; j++) {
			dot += P_apriori[i + j * N] * H[j];
		}
		P_H[i] = dot;
	}
	for(int i = 0; i < N; i++) {
		s += H[i] * P_H[i];
	}

#ifdef DEBUG
	if(obs == 1 && tid == 0) {
		printf("s %.2lf\n", s);
	}
#endif

	// kalman gain

	auto K = P_H + N;
	for(int i = tid; i < N; i += block_dim) {
		K[i] = P_H[i] / s;
	}

	////////////
	// update //
	////////////

	// state update

	for(int i = tid; i < N; i += block_dim) {
		x[i] += y * K[i];
	}

	// covariance update

	auto diff = K + N;
	for(int j = tid; j < N; j += block_dim) {
		for(int i = 0; i < N; i++) {
			diff[i + j * N] = (i == j ? 1 : 0) - K[i] * H[j];
		}
	}
	for(int j = tid; j < N; j += block_dim) {
		for(int i = 0; i < N; i++) {
			double dot = 0;
			for(int k = 0; k < N; k++) {
				dot += diff[i + k * N] * P_apriori[k + j * N];
			}
			P[i + j * N] = dot;
		}
	}

#ifdef DEBUG
    if(obs == 1 && tid == 0) {
        printf("\n\n");
    }
#endif

	return KalmanResult{y, s};
}

__global__ void kalman_cuda(int N, double *d_x, double *d_P, double *d_prices,
                            char *d_positions, double *d_scratch) {
#ifdef DEBUG
	printf("kalman_cuda called......\n");
#endif

	auto N2 = N * N;
	auto grid_dim = gridDim.x;
	auto block_idx = blockIdx.x;
	auto scratch = d_scratch + (block_idx * SCRATCH_SIZE);

	for(int obs = block_idx; obs < N; obs += grid_dim) {
		auto x = d_x + (obs * N);
		auto P = d_P + (obs * N2);
		auto result = kalman_cuda_update(N, obs, x, P, d_prices, scratch);
	}

#ifdef DEBUG
	printf("kalman_cuda exited......\n");
#endif
}

unordered_set<string> FLAGS{};
unordered_set<string> OPTS{"-i", "-o", "-g", "-b"};

int main(int argc, char **argv) {
	auto args = parse_args(argc, argv, FLAGS, OPTS);

	auto ifile = fopen(args["-i"].c_str(), "r");
	auto ofile = fopen(args["-o"].c_str(), "w");
	fscanf(ifile, "%d %d", &N, &days);

	grid_dim = stoi(args["-g"]);
	block_dim = stoi(args["-b"]);

	// init memory regions

	double *h_prices;
	hipHostMalloc(&h_prices, N * sizeof(double));
	char *h_positions;
	hipHostMalloc(&h_positions, N);

	double *d_x;
	hipMalloc(&d_x, N * N * sizeof(double));
	double *d_P;
	hipMalloc(&d_P, N * N * N * sizeof(double));
	double *d_prices;
	hipMalloc(&d_prices, N * sizeof(double));
	char *d_positions;
	hipMalloc(&d_positions, N);
	double *d_scratch;
	hipMalloc(&d_scratch, grid_dim * SCRATCH_SIZE * sizeof(double));

	auto h_x = (double *)malloc(N * N * sizeof(double));
	auto h_P = (double *)malloc(N * N * N * sizeof(double));
	for(int obs = 0; obs < N; obs++) {
		auto x = h_x + obs * N;
		auto P = h_P + obs * N * N;
		for(int i = 0; i < N; i++) {
			x[i] = 0;
			P[i + i * N] = P0;
		}
	}
	hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_P, h_P, N * N * sizeof(double), hipMemcpyHostToDevice);
	free(h_x);
	free(h_P);

	// do kalman

	for(int d = 0; d < days; d++) {
		printf("day %d\n", d);

		// refresh prices

		double price;
		for(int i = 0; i < N; i++) {
			fscanf(ifile, "%lf", &price);
			h_prices[i] = price;
		}

		// kalman update

		auto t0 = chrono::system_clock::now();
		long elapsed;

		hipMemcpy(d_prices, h_prices, N * sizeof(double),
		           hipMemcpyHostToDevice);

		kalman_cuda<<< grid_dim, block_dim >>>(N, d_x, d_P, d_prices,
		                                       d_positions, d_scratch);
		hipDeviceSynchronize();

		hipMemcpy(h_positions, d_positions, N, hipMemcpyDeviceToHost);

		auto t1 = chrono::system_clock::now();
		elapsed = (long)((t1 - t0) / chrono::microseconds(1));
		printf("kalman %ld\n", elapsed);
	}

	hipHostFree(h_prices);
	hipHostFree(h_positions);
	hipFree(d_x);
	hipFree(d_P);
	hipFree(d_prices);
	hipFree(d_positions);
	hipFree(d_scratch);

	fclose(ifile);
	fclose(ofile);

	return 0;
}
