#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "argparse.h"
#include "kalman.h"

//#define DEBUG

using namespace std;

#define SCRATCH_SIZE (3 * N * N + 4 * N)

int N, days, grid_dim, block_dim;

__device__ KalmanResult kalman_cuda_update(int N, int obs,
                                           double *x, double *P,
                                           const double *prices,
                                           double *scratch) {
	auto tid = threadIdx.x;
	auto block_dim = blockDim.x;

	auto N2 = N * N;
	auto Q = DELTA / (1 - DELTA);
	auto z = prices[obs];
	auto H = scratch + 0;
	int i_H = 0;
	for(int i = 0; i < N; i++) {
		if(i == obs)
			continue;
		H[i_H] = prices[i];
		i_H++;
	}
	H[i_H] = 1;

#ifdef DEBUG
	if(obs == 1 && tid == 0) {
		printf("H\n");
		for(int i = 0; i < N; i++) {
			printf("%.2lf\t", H[i]);
		}
		printf("\n");
	}
#endif

	/////////////
	// predict //
	/////////////

	// state prediction

	auto x_apriori = x;

#ifdef DEBUG
	if(obs == 1 && tid == 0) {
		printf("x_apriori\n");
		for(int i = 0; i < N; i++) {
			printf("%.2lf\t", x_apriori[i]);
		}
		printf("\n");
	}
#endif

	// state covariance prediction

	auto P_apriori = H + N;
	for(int j = tid; j < N; j += block_dim) {
		for(int i = 0; i < N; i++) {
			auto idx = i + j * N;
			P_apriori[idx] = P[idx];
			if(i == j)
				P_apriori[idx] += Q;
		}
	}

#ifdef DEBUG
    if(obs == 1 && tid == 0) {
        printf("P_apriori\n");
        for(int i = 0; i < N; i++) {
            for(int j = 0; j < N; j++) {
                printf("%.2lf\t", P_apriori[i + j * N]);
            }
            printf("\n");
        }
    }
#endif

	///////////////
	// calculate //
	///////////////

	// innovation residual

	double y = z;
	for(int i = 0; i < N; i++) {
		y -= H[i] * x_apriori[i];
	}

#ifdef DEBUG
	if(obs == 1 && tid == 0) {
		printf("y %.2lf\n", y);
	}
#endif

	// innovation covariance

	double s = R;
	auto P_H = P_apriori + N2;
	for(int i = tid; i < N; i += block_dim) {
		double dot = 0;
		for(int j = 0; j < N; j++) {
			dot += P_apriori[i + j * N] * H[j];
		}
		P_H[i] = dot;
	}
	for(int i = 0; i < N; i++) {
		s += H[i] * P_H[i];
	}

#ifdef DEBUG
	if(obs == 1 && tid == 0) {
		printf("s %.2lf\n", s);
	}
#endif

	// kalman gain

	auto K = P_H + N;
	for(int i = tid; i < N; i += block_dim) {
		K[i] = P_H[i] / s;
	}

	////////////
	// update //
	////////////

	// state update

	for(int i = tid; i < N; i += block_dim) {
		x[i] += y * K[i];
	}

	// covariance update

	auto diff = K + N;
	for(int j = tid; j < N; j += block_dim) {
		for(int i = 0; i < N; i++) {
			diff[i + j * N] = (i == j ? 1 : 0) - K[i] * H[j];
		}
	}
	for(int j = tid; j < N; j += block_dim) {
		for(int i = 0; i < N; i++) {
			double dot = 0;
			for(int k = 0; k < N; k++) {
				dot += diff[i + k * N] * P_apriori[k + j * N];
			}
			P[i + j * N] = dot;
		}
	}

#ifdef DEBUG
    if(obs == 1 && tid == 0) {
        printf("\n\n");
    }
#endif

	return KalmanResult{y, s};
}

__global__ void kalman_cuda(int N, double *d_prices, char *d_positions,
                            double *d_scratch) {
#ifdef DEBUG
	printf("kalman_cuda called......\n");
#endif

	auto N2 = N * N;
	auto grid_dim = gridDim.x;

	for(int obs = blockIdx.x; obs < N; obs += grid_dim) {
		auto scratch_off = obs * SCRATCH_SIZE;
		auto x = d_scratch + scratch_off;
		auto P = x + N;
		auto prices = d_prices;
		auto scratch = P + N2;
		auto result = kalman_cuda_update(N, obs, x, P, prices, scratch);
	}

#ifdef DEBUG
	printf("kalman_cuda exited......\n");
#endif
}

unordered_set<string> FLAGS{};
unordered_set<string> OPTS{"-i", "-o", "-g", "-b"};

int main(int argc, char **argv) {
	auto args = parse_args(argc, argv, FLAGS, OPTS);

	auto ifile = fopen(args["-i"].c_str(), "r");
	auto ofile = fopen(args["-o"].c_str(), "w");
	fscanf(ifile, "%d %d", &N, &days);

	// read price series

	auto h_prices = (double *)calloc(days * N, sizeof(double));
	for(int d = 0; d < days; d++) {
		double price;
		for(int i = 0; i < N; i++) {
			fscanf(ifile, "%lf", &price);
			h_prices[d * N + i] = price;
		}
	}

	// init memory regions
	
	double *d_prices;
	hipMalloc(&d_prices, N * sizeof(double));
	char *d_positions;
	hipMalloc(&d_positions, N);
	double *d_scratch;
	hipMalloc(&d_scratch, N * SCRATCH_SIZE * sizeof(double));

	auto h_scratch = (double *)malloc(N * SCRATCH_SIZE * sizeof(double));
	for(int obs = 0; obs < N; obs++) {
		auto scratch_off = obs * SCRATCH_SIZE;
		auto x = h_scratch + scratch_off;
		auto P = x + N;
		for(int i = 0; i < N; i++) {
			x[i] = 0;
			P[i + i * N] = P0;
		}
	}
	hipMemcpy(d_scratch, h_scratch, N * SCRATCH_SIZE * sizeof(double),
	           hipMemcpyHostToDevice);

	// do kalman

	grid_dim = stoi(args["-g"]);
	block_dim = stoi(args["-b"]);

	for(int d = 0; d < days; d++) {
		printf("day %d\n", d);

		auto t0 = chrono::system_clock::now();
		long elapsed;

		hipMemcpy(d_prices, h_prices + d * N, N * sizeof(double),
		           hipMemcpyHostToDevice);

		auto t1 = chrono::system_clock::now();
		elapsed = (long)((t1 - t0) / chrono::microseconds(1));
		printf("memcpy %ld\n", elapsed);

		kalman_cuda<<< grid_dim, block_dim >>>(N, d_prices, d_positions,
		                                       d_scratch);
		hipDeviceSynchronize();

		auto t2 = chrono::system_clock::now();
		elapsed = (long)((t2 - t1) / chrono::microseconds(1));
		printf("kalman %ld\n", elapsed);
	}

	// print

	// printf("total P&L: %.4lf\n", total_pnl);
	// printf("%.4lf\n", total_pnl);

	fclose(ifile);
	fclose(ofile);

	return 0;
}
