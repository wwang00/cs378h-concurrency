#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <stdio.h>
#include <vector>

#include "argparse.h"
#include "kalman.h"

#define DEBUG

using namespace std;

#define GRID_DIM 1
#define BLOCK_DIM 1

int stonks, days, tests;
int data_bytes, out_bytes, scratch_bytes;

double *h_prices, *h_pnl;
double *d_prices, *d_pnl;
double *scratch; // workspace for cuda routine

// first entry: initial prices
// following entries: changes in prices
vector<vector<double>> raw_price_data;

__device__ KalmanResult kalman_cuda_update(const int N, const int obs, double *x, double *P,
                                           const double *prices, double *scratch) {
	auto N2 = N * N;
	auto Q = DELTA / (1 - DELTA);
	auto z = prices[obs];
	auto H = scratch + 0;
	int i_H = 0;
	for(int i = 0; i < N; i++) {
		if(i == obs)
			continue;
		H[i_H] = prices[i];
		i_H++;
	}
	H[i_H] = 1;


	/////////////
	// predict //
	/////////////

	// state prediction

	auto x_apriori = x;

	// state covariance prediction

	auto P_apriori = H + N;
	for(int j = 0; j < N; j++) {
		for(int i = 0; i < N; i++) {
			auto idx = i + j * N;
			P_apriori[idx] = P[idx];
			if(i == j)
				P_apriori[idx] += Q;
		}
	}

	///////////////
	// calculate //
	///////////////

	// innovation residual

	double y = z;
    for(int i = 0; i < N; i++) {
	    y -= H[i] * x_apriori[i];
    }

	// innovation covariance

	double s = R;
	auto P_H = P_apriori + N2;
    for(int i = 0; i < N; i++) {
	    double dot = 0;
	    for(int j = 0; j < N; j++) {
		    dot += P_apriori[i + j * N] * H[j];
	    }
	    P_H[i] = dot;
    }
    for(int i = 0; i < N; i++) {
	    s += H[i] * P_H[i];
    }

    // kalman gain

    auto K = P_H + N;
    for(int i = 0; i < N; i++) {
	    K[i] = P_H[i] / s;
    }

	////////////
	// update //
	////////////

	// state update

    for(int i = 0; i < N; i++) {
	    x[i] += y * K[i];
    }

    // covariance update

    auto diff = K + N;
	for(int j = 0; j < N; j++) {
		for(int i = 0; i < N; i++) {
			diff[i + j * N] = (i == j ? 1 : 0) - K[i] * H[j];
		}
	}
	for(int j = 0; j < N; j++) {
		for(int i = 0; i < N; i++) {
			double dot = 0;
			for(int k = 0; k < N; k++) {
				dot += diff[i + k * N] * P_apriori[k + j * N];
			}
			P[i + j * N] = dot;
		}
	}

	return KalmanResult{y, s};
}

__device__ void kalman_cuda(int N, int days, const double *prices, double *pnl, double *scratch) {
	printf("kalman_cuda called......\n");

	auto N2 = N * N;

	auto *x = scratch + 0;
	auto *P = x + N;
	for(int j = 0; j < N; j++) {
		x[j] = 0;
		for(int i = 0; i < N; i++) {
			P[i + j * N] = i == j ? P0 : 0;
		}
	}

	int position = 0;
	double exit_zscore;
	double last_beta;
	double last_port;
	double total_pnl = 0;
	int total_trades = 0;
	for(int d = 0; d < days; d++) {
		auto px = prices[d * N];
		auto py = prices[d * N + 1];
		auto beta = x[0];
		auto intc = x[1];
		auto result = kalman_cuda_update(N, 1, x, P, &prices[d * N], P + N2);
		if(d < TRAINING_DAYS) {
			pnl[d] = 0;
			continue;
		}
		int sgn = result.y < 0 ? -1 : 1;
		auto zscore = sgn * result.y * result.y / (result.s - R);

		if(zscore < -ZSCORE_ENTRY && position == 0) {
			exit_zscore = zscore + ZSCORE_DELTA;
			last_beta = beta;
			last_port = py - px * beta;
			total_trades++;
			position = 1;
		}
		if(zscore > exit_zscore && position == 1) {
			auto port = py - px * last_beta;
			total_pnl += port - last_port;
			total_trades++;
			position = 0;
		}
		if(zscore > ZSCORE_ENTRY && position == 0) {
			exit_zscore = zscore - ZSCORE_DELTA;
			last_beta = beta;
			last_port = py - px * beta;
			total_trades++;
			position = -1;
		}
		if(zscore < exit_zscore && position == -1) {
			auto port = py - px * last_beta;
			total_pnl += last_port - port;
			total_trades++;
			position = 0;
		}

		pnl[d] = total_pnl;
	}
	printf("kalman_cuda exited......\n");
}

__global__ void ExecuteStrategy(int tests, int stonks, int days, double *prices, double *pnl, double *scratch) {
	printf("ExecuteStrategy called......\n");

    // check bounds
    int test_id = blockDim.x * blockIdx.x + threadIdx.x;
    if(test_id >= tests) return;

    // flat array
    int base = test_id * stonks * days;
	kalman_cuda(stonks, days, &prices[base], &pnl[base], &scratch[base]);

	printf("ExecuteStrategy exited......\n");
}

void load_data(string filename) {
#ifdef DEBUG
	printf("load_data called......\n");
#endif
	// load data
	ifstream fin(filename);
	fin >> stonks >> days;

	// size of all data
	data_bytes = stonks * days * tests * sizeof(double);

	raw_price_data.resize(days, vector<double>(stonks));
	vector<double> last_prices(stonks);
	for(int i = 0; i < days; i++) {
		for(int j = 0; j < stonks; j++) {
			double price;
			fin >> price;
			if(i == 0) {
				raw_price_data[i][j] = price;
			} else {
				raw_price_data[i][j] = price - last_prices[j];
			}
			last_prices[j] = price;
		}
	}
	fin.close();

#ifdef DEBUG
	for(int i = 0; i < days; i++) {
		for(int j = 0; j < stonks; j++) {
			printf("%.4lf\t", raw_price_data[i][j]);
		}
		printf("\n");
	}
	printf("load_data exited......\n");
#endif
}

void gen_data() {
#ifdef DEBUG
	printf("gen_data called......\n");
#endif
	h_prices = (double *)malloc(data_bytes);
	for(int t = 0; t < tests; t++) {
		// shuffle whole day arrays
		random_shuffle(++raw_price_data.begin(), raw_price_data.end());

		// copy and accumulate shuffled arrays
		for(int i = 0; i < days; i++) {
			int off = stonks * (t * days + i);
			for(int j = 0; j < stonks; j++) {
				if(i == 0) {
					h_prices[off + j] = raw_price_data[i][j];
				} else {
					h_prices[off + j] =
					    raw_price_data[i][j] + h_prices[off - stonks + j];
				}
			}
		}
	}
#ifdef DEBUG
	for(int t = 0; t < tests; t++) {
		printf("test %d\n", t);
		for(int i = 0; i < days; i++) {
			printf("\tday %d\n", i);
			int off = stonks * (t * days + i);
			printf("\t\t");
			for(int j = 0; j < stonks; j++) {
				printf("%.4lf\t", h_prices[off + j]);
			}
			printf("\n");
		}
	}

	printf("gen_data exited......\n");
#endif
}

void backtest() {
	printf("backtest called......\n");

	// copy to device
	hipMalloc(&d_prices, data_bytes);
	hipMemcpy(d_prices, h_prices, data_bytes,
	           hipMemcpyHostToDevice);

	out_bytes = tests * days * sizeof(double);
	// output array
	h_pnl = (double *)malloc(out_bytes);
	hipMalloc(&d_pnl, out_bytes);

	scratch_bytes = tests * (3 * stonks * stonks + 4 * stonks) * sizeof(double);
	hipMalloc(&scratch, scratch_bytes);

	// execute strategy
	ExecuteStrategy<<<GRID_DIM, BLOCK_DIM>>>(tests, stonks, days, d_prices, d_pnl, scratch);
	hipDeviceSynchronize();

	// device -> host
	hipMemcpy(h_pnl, d_pnl, out_bytes, hipMemcpyDeviceToHost);

	for(int t = 0; t < tests; t++) {
		int off = t * days;
		printf("Test %d - total P&L: %.4lf\n", t, h_pnl[off + days - 1]);
	}

	printf("backtest exited......\n");
	return;
}

unordered_set<string> FLAGS{};
unordered_set<string> OPTS{"-i", "-t"};

int main(int argc, char **argv) {
	auto args = parse_args(argc, argv, FLAGS, OPTS);
	auto filename = args["-i"];
	tests = stoi(args["-t"]);

	load_data(filename);
	gen_data();
	backtest();

	return 0;
}
